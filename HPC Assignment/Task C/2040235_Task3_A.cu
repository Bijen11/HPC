
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>





__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;  
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;  
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){//checking all lower case letter and numbers limits
		if(i >= 0 && i < 6){ 
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ 
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword; //Returns encrypted password
}

__device__ int compare(char* stringOne, char* stringTwo){
	
    while(*stringOne)
    {
        
        if (*stringOne != *stringTwo)
            break;
 
        //increasing Pointer location
        stringOne++;
        stringTwo++;
    }
 
    
    return *(const unsigned char*)stringOne - *(const unsigned char*)stringTwo;
}

__global__ void crackpass(char * alphabet, char * numbers, char * rawPassword){

char genRawPass[4];

genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];

genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];


char *encPassword = CudaCrypt(rawPassword);
	
	//Comparing encrypted genRawPass with encPassword
	if(compare(CudaCrypt(genRawPass),encPassword) == 0){
		printf("Your password is cracked : %s = %s\n", encPassword, rawPassword);
	}
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
	  long long int ds =  finish->tv_sec - start->tv_sec; 
	  long long int dn =  finish->tv_nsec - start->tv_nsec; 

	  if(dn < 0 ) 
	  {
	    ds--;
	    dn += 1000000000; 
          } 

	  *difference = ds * 1000000000 + dn;
	  return !(*difference > 0);
}

int main(int argc, char ** argv){

char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

char * gpuAlphabet;
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

char * password;
hipMalloc( (void**) &password, sizeof(char) * 26); 
hipMemcpy(password, argv[1], sizeof(char) * 26, hipMemcpyHostToDevice);

	struct timespec start, finish;
	long long int time_elapsed;
	

	clock_gettime(CLOCK_MONOTONIC, &start);//Start monitoring the duration 
	
	crackpass<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers, password);
	hipDeviceSynchronize();


	clock_gettime(CLOCK_MONOTONIC, &finish); //End the duration of the program
	
//Calculate difference
	time_difference(&start, &finish, &time_elapsed);
	

	printf(" Time taken to crack : %lld ns \n",time_elapsed); //Print the duration taken
return 0;
}


	











